#include "hip/hip_runtime.h"
// 1. Write a program in CUDA to count the number of times a given word is repeated in a sentence.
// (Use Atomic function)
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ count_occur_kernel(char* text, char* word, int* count,int lt,int lw)
{
    int i= threadIdx.x + blockIdx.x*blockDim.x;
    if(i<lt-lw)
    {
        int check=1;
        for(int  j=0;j<lw && text[i+j]==word[j];j++);
        {
            if(text[i+j]!=word[j])
            {
                check=0;break;
            }
        }
        if(check)
            atomicAdd(count,1);
    }
}

int count_occur(char* text, char* word)
{
    int lt=strlen(text),lw=strlen(word),size=sizeof(int);
    int* count=0,*d_count;
    char* d_text;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); 

    hipMalloc((void **)&d_text, lt * sizeof(char));
    hipMalloc((void **)&d_count, 1 * size);
    hipMemcpy(d_text, text, lt * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, count, 1 * size, hipMemcpyHostToDevice);

    dim3 dimGrid(n, 1, 1);
    dim3 dimBlock(1, 1, 1);
    count_occur_kernel<<<dimGrid, dimBlock>>>(d_text, word, d_count, lt,lw);
    error =hipGetLastError();
    if (error != hipSuccess) 
        printf("CUDA Error2: %s\n", hipGetErrorString(error));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsed Time,start, stop); 
    printf("Time Taken=%f" elapsed Time); 


    hipMemcpy(count, d_count, 1 * size, hipMemcpyDeviceToHost);

    hipFree(d_text);
    hipFree(d_count);
    return *count;
}
int main()
{
    char text[100], word[10];
    int lt,lw;
    printf("Enter the sentence");
    scanf("%s",text);
    lt=strlen(text);
    printf("Enter the word to be searched");
    scanf("%s",word);
    lw=strlen(word);
    int result = count_occur(text,word);
    printf("Total no. of occurrences of %s in text is= %d", text, result);
    return 0;



}