#include "hip/hip_runtime.h"
// 1. Write a program in CUDA to perform parallel Sparse Matrix - Vector multiplication using com-
// pressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format in the
// host code.
#include <stdio.h>
#include <hip/hip_runtime.h>
int m, n;
#define cc 4
__global__ void csr_spmv(int m, int *row_ptr, int *col_idx, float *values, float *x, float *y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m)
    {
        float sum = 0.0f;
        int start = row_ptr[row];
        int end = row_ptr[row + 1];
        for (int i = start; i < end; i++)
        {
            sum += values[i] * x[col_idx[i]];
        }
        y[row] = sum;
    }
}

void convert_to_csr(int m, int n, float dense_matrix[][cc],
                    int *&row_ptr, int *&col_idx, float *&values, int &nnz)
{
    nnz = 0;
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            if (dense_matrix[i][j] != 0)
                nnz++;

    row_ptr = new int[m + 1];
    col_idx = new int[nnz];
    values = new float[nnz];

    int current = 0;
    row_ptr[0] = 0;

    for (int i = 0; i < m; i++)
    {
        int row_nnz = 0;
        for (int j = 0; j < n; j++)
        {
            if (dense_matrix[i][j] != 0)
            {
                col_idx[current] = j;
                values[current] = dense_matrix[i][j];
                current++;
                row_nnz++;
            }
        }
        row_ptr[i + 1] = row_ptr[i] + row_nnz;
    }
}

int main()
{
    m = 4, n = 4;
    float dense_matrix[][cc] = {
        {1.0, 2.0, 0.0, 0.0},
        {0.0, 0.0, 3.0, 0.0},
        {0.0, 4.0, 0.0, 5.0},
        {0.0, 0.0, 6.0, 0.0}};

    int *row_ptr = nullptr, *col_idx = nullptr;
    float *values = nullptr;
    int nnz;
    convert_to_csr(m, n, dense_matrix, row_ptr, col_idx, values, nnz);

    float x[] = {1.0, 2.0, 3.0, 4.0};
    float y[m] = {0.0};

    int *d_row_ptr, *d_col_idx;
    float *d_values, *d_x, *d_y;

    hipMalloc(&d_row_ptr, (m + 1) * sizeof(int));
    hipMalloc(&d_col_idx, nnz * sizeof(int));
    hipMalloc(&d_values, nnz * sizeof(float));
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, m * sizeof(float));

    hipMemcpy(d_row_ptr, row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (m + 31) / 32;
    csr_spmv<<<blocks, 32>>>(m, d_row_ptr, d_col_idx, d_values, d_x, d_y);

    hipMemcpy(y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

    printf("Result vector:\n");
    for (int i = 0; i < m; i++)
    {
        printf("y[%d] = %.2f\n", i, y[i]);
    }

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
// Result vector:
// y[0] = 5.00
// y[1] = 9.00
// y[2] = 28.00
// y[3] = 18.00