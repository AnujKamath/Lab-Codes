// Write a program in CUDA to process a 1D array containing angles in radians to generate
// sine of the angles in the output array. Use appropriate function.

#include "hip/hip_runtime.h"

#include <stdio.h>

#include <math.h>
__global__ void vecSineKernel(float *a, float *b, int n)
{
    size_t i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
        b[i] = sinf(a[i]);
}
void vecSine(float *a, float *b, int n)
{
    float *d_a, *d_b;
    int size = sizeof(float);
    hipMalloc((void **)&d_a, n * size);
    hipMalloc((void **)&d_b, n * size);

    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(n / 256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);
    vecSineKernel<<<dimGrid, dimBlock>>>(d_a, d_b, n);

    hipMemcpy(b, d_b, n * size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
}
int main()
{
    int n = 10;
    float a[n], b[n];
    for (int i = 0, c = 0; i < n; i++, c += 15)
    {
        a[i] = (c * M_PI) / 180;
    }
    printf("Radians: ");

    for (int i = 0; i < n; i++)
        printf("%.2f ", a[i]);
    printf("\n");

    vecSine(a, b, n);
    printf("Result : ");

    for (int i = 0; i < n; i++)
        printf("%.2f ", b[i]);
    printf("\n");

    return 0;
}
// student@dbl-34:~/Documents/220962446_PCAP/Lab5$ ./out1
// Radians: 0.00 0.26 0.52 0.79 1.05 1.31 1.57 1.83 2.09 2.36
// Result : 0.00 0.26 0.50 0.71 0.87 0.97 1.00 0.97 0.87 0.71
