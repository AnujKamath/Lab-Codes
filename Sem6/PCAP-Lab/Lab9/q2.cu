// Question 2. Write a program in CUDA to read MXN matrix A and replace 1% row of this matrix by same
// elements, 2" row elements by square of each element and 3™ row elements by cube of each element
// and so on.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
__global__ void computeRowExponents(int *matrix, int M, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N)
    {
        matrix[row * N + col] = (int)pow((float)matrix[row * N + col], (float)row + 1);
    }
}

int main()
{
    const int M = 3;
    const int N = 3;

    int h_matrix[M][N] = {
        {1, 2, 3},
        {4, 5, 6},
        {7, 8, 9}};

    int *d_matrix;
    hipMalloc((void **)&d_matrix, M * N * sizeof(int));

    hipMemcpy(d_matrix, h_matrix, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(1, 1);
    dim3 gridSize(N, M);

    computeRowExponents<<<gridSize, blockSize>>>(d_matrix, M, N);

    hipMemcpy(h_matrix, d_matrix, M * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Modified matrix:\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d\t", h_matrix[i][j]);
        }
        printf("\n");
    }

    hipFree(d_matrix);

    return 0;
}
// Modified matrix:
// 1       2       2
// 16      25      36
// 343     512     729