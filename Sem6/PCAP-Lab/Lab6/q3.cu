// Write a program in CUDA to perform odd even transposition sort in parallel.
#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void odd_sort_kernel(int *num, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n - 1 && i % 2)
    {
        if (num[i] > num[i + 1])
        {
            int temp = num[i];
            num[i] = num[i + 1];
            num[i + 1] = temp;
        }
    }
}
__global__ void even_sort_kernel(int *num, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n - 1 && i % 2 == 0)
    {
        if (num[i] > num[i + 1])
        {
            int temp = num[i];
            num[i] = num[i + 1];
            num[i + 1] = temp;
        }
    }
}
void oe_sort(int *num, int n)
{
    int *d_num;
    int size = sizeof(int);
    hipMalloc((void **)&d_num, n * size);

    hipMemcpy(d_num, num, n * size, hipMemcpyHostToDevice);

    dim3 dimGrid(n, 1, 1);
    dim3 dimBlock(1, 1, 1);
    for (int i = 0; i < n / 2; i++)
    {
        odd_sort_kernel<<<dimGrid, dimBlock>>>(d_num, n);
        even_sort_kernel<<<dimGrid, dimBlock>>>(d_num, n);
    }
    hipMemcpy(num, d_num, n * size, hipMemcpyDeviceToHost);

    hipFree(d_num);
}
int main()
{
    int n;
    printf("Enter size of array: ");
    scanf("%d", &n);
    int num[n];
    printf("\nEnter values of array N: ");
    for (int i = 0; i < n; i++)
    {
        scanf("%d", &num[i]);
    }
    oe_sort(num, n);
    printf("Sorted Array: ");
    for (int i = 0; i < n; i++)
        printf("%d ", num[i]);
    printf("\n");

    return 0;
}
// student@dbl-34:~/Documents/220962446_PCAP/Lab6$ ./out1
// Enter size of array: 6

// Enter values of array N: 4 3 5 6 2
// 1
// Sorted Array: 1 2 3 4 5 6