#include "hip/hip_runtime.h"
// Write a program in CUDA to perform tiled 1D convolution operation on the input array N of
// size width using the mask array, M of size mask_width, to produce the resultant array P of size
// width.
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 256
#define MASK_WIDTH 5

__global__ void tiled1DConvolution(float *N, float *M, float *P, int width, int mask_width)
{
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int half_mask = mask_width / 2;
    __shared__ float tile[TILE_SIZE + MASK_WIDTH - 1];
    int tile_start = blockIdx.x * blockDim.x;
    int local_index = threadIdx.x;
    int g_load = tile_start + local_index - half_mask;

    tile[local_index] = (g_load >= 0 && g_load < width) ? N[g_load] : 0.0f;

    if (threadIdx.x < mask_width - 1 && blockIdx.x < gridDim.x - 1)
    {
        int next_g_load = tile_start + blockDim.x + threadIdx.x - half_mask;
        tile[blockDim.x + threadIdx.x] = (next_g_load >= 0 && next_g_load < width) ? N[next_g_load] : 0.0f;
    }
    if (threadIdx.x >= blockDim.x - (mask_width - 1) && blockIdx.x > 0)
    {
        int prev_g_load = tile_start - (blockDim.x - threadIdx.x) - half_mask;
        tile[threadIdx.x - (blockDim.x - (mask_width - 1))] = (prev_g_load >= 0 && prev_g_load < width) ? N[prev_g_load] : 0.0f;
    }
    __syncthreads();

    if (global_index < width)
    {
        float sum = 0.0f;
        for (int k = 0; k < mask_width; ++k)
            sum += tile[local_index + k] * M[k];
        P[global_index] = sum;
    }
}

int main()
{
    int width = 7;
    int mask_width = MASK_WIDTH;

    float *d_N, *d_M, *d_P;
    float h_N[width] = {1, 2, 3, 4, 5, 6, 7};
    float h_M[mask_width] = {3, 4, 5, 4, 3};
    float h_P[width];
    hipMalloc((void **)&d_N, width * sizeof(float));
    hipMalloc((void **)&d_M, mask_width * sizeof(float));
    hipMalloc((void **)&d_P, width * sizeof(float));
    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);
    int tpb = TILE_SIZE;
    int nb = (width + tpb - 1) / tpb;
    tiled1DConvolution<<<nb, tpb>>>(d_N, d_M, d_P, width, mask_width);
    hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

    printf("Input Array N:\n");
    for (int i = 0; i < width; ++i)
        printf("%.2f ", h_N[i]);
    printf("\n");

    printf("Mask M:\n");
    for (int i = 0; i < mask_width; ++i)
        printf("%.2f ", h_M[i]);
    printf("\n");

    printf("Resultant Array P:\n");
    for (int i = 0; i < width; ++i)
        printf("%.2f ", h_P[i]);
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
    return 0;
}

// Input Array N:
// 1.00 2.00 3.00 4.00 5.00 6.00 7.00
// Mask M:
// 3.00 4.00 5.00 4.00 3.00
// Resultant Array P:
// 22.00 38.00 57.00 76.00 95.00 90.00 74.00