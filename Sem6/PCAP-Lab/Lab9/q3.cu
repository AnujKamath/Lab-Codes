// Question 3. Write a CUDA program that reads a matrix A of size MXN and produce an output matrix B of
// same size such that it replaces all the non-border elements (numbers in bold) of A with its equivalent
// 1’s complement and remaining elements same as matrix A.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
__device__ int complement(int val)
{
    int ans = 0, exp = 1;
    for (int i = val; i > 0; i /= 2, exp *= 10)
    {
        ans += (1 - (i % 2)) * exp;
    }
    return ans;
}
__global__ void computeRowExponents(int *in_mat, int *out_mat, int M, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N)
    {
        if (row == 0 || row == M - 1 || col == 0 || col == N - 1)
            out_mat[row * N + col] = in_mat[row * N + col];
        else
            out_mat[row * N + col] = complement(in_mat[row * N + col]);
    }
}

int main()
{
    int M, N;
    printf("Enter the dimension of matrix");
    scanf("%d %d", &M, &N);
    int matrix[M][N];
    printf("Enter %d elements", M * N);
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++)
            scanf("%d", &matrix[i][j]);
    int output[M][N];

    int *d_matrix, *d_output;
    hipMalloc((void **)&d_matrix, M * N * sizeof(int));
    hipMalloc((void **)&d_output, M * N * sizeof(int));

    hipMemcpy(d_matrix, matrix, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(1, 1);
    dim3 gridSize(N, M);

    computeRowExponents<<<gridSize, blockSize>>>(d_matrix, d_output, M, N);

    hipMemcpy(output, d_output, M * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Modified matrix:\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d\t", output[i][j]);
        }
        printf("\n");
    }

    hipFree(d_matrix);
    hipFree(d_output);
    return 0;
}
// Enter the dimension of matrix: 4 4
// 1 2 3 4 6 5 8 3 2 4 10 1 9 1 2 5
// Modified matrix:
// 1       2       3       4
// 6       10      111     3
// 2       11      101     1
// 9       1       2       5