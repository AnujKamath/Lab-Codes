// Question 1 Write a program in CUDA to perform matrix multiplication using 2D Grid and 2D Block.

#include "hip/hip_runtime.h"

#include <stdio.h>

#define WIDTH 2
__global__ void kernel_c(int *A, int *B, int *C)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < WIDTH && col < WIDTH)
    {
        int x = 0;
        for (int k = 0; k < WIDTH; k++)
            x += A[row * WIDTH + k] * B[k * WIDTH + col];
        C[row * WIDTH + col] = x;
    }
}
void mul_matrix(int A[][WIDTH], int B[][WIDTH])
{
    int C[WIDTH][WIDTH], size = sizeof(int);
    int *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, WIDTH * WIDTH * size);
    hipMalloc((void **)&d_B, WIDTH * WIDTH * size);
    hipMalloc((void **)&d_C, WIDTH * WIDTH * size);

    hipMemcpy(d_A, A, WIDTH * WIDTH * size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, WIDTH * WIDTH * size, hipMemcpyHostToDevice);

    kernel_c<<<dim3(ceil(WIDTH / 32.0), ceil(WIDTH / 32.0)), dim3(32, 32)>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, WIDTH * WIDTH * size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Resultant array:\n");
    for (int i = 0; i < WIDTH; i++)
    {
        for (int j = 0; j < WIDTH; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }
}
int main()
{

    int A[WIDTH][WIDTH] = {{1, 2}, {2, 3}};
    int B[WIDTH][WIDTH] = {{3, 4}, {5, 6}};
    mul_matrix(A, B);
    return 0;
}
// Resultant array:
// 13 16
// 21 26