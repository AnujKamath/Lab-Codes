#include "hip/hip_runtime.h"
// Question 1. Write a progrm in CUDA to add two Matrices for the following specifications:
// a.Each row of resultant matrix to be computed by one thread.
// b.Each column of resultant matrix to be computed by one thread.
// c.Each element of resultant matrix to be computed by one thread.

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#define COL 3
__global__ void kernel_a(int *A, int *B, int *C, int hA, int wA, int wB)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < hA)
    {
        for (int col = 0; col < wB; col++)
        {
            int x = 0;
            for (int k = 0; k < wA; k++)
            {
                x += A[row * wA + k] * B[k * wB + col];
            }
            C[row * wB + col] = x;
        }
    }
}
__global__ void kernel_b(int *A, int *B, int *C, int hA, int wA, int wB)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (col < wB)
    {
        for (int row = 0; row < hA; row++)
        {
            int x = 0;
            for (int k = 0; k < wA; k++)
            {
                x += A[row * wA + k] * B[k * wB + col];
            }
            C[row * wB + col] = x;
        }
    }
}
__global__ void kernel_c(int *A, int *B, int *C, int hA, int wA, int wB)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < hA && col < wB)
    {
        int x = 0;
        for (int k = 0; k < wA; k++)
            x += A[row * wA + k] * B[k * wB + col];
        C[row * wB + col] = x;
    }
}
void mul_matrix(int A[][COL], int B[][COL], int hA, int wA, int wB, int opt)
{
    int C[hA][wB], size = sizeof(int);
    int *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, hA * wA * size);
    hipMalloc((void **)&d_B, wA * wB * size);
    hipMalloc((void **)&d_C, hA * wB * size);

    hipMemcpy(d_A, A, hA * wA * size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, wA * wB * size, hipMemcpyHostToDevice);

    if (opt == 1)
        kernel_a<<<ceil(hA / 32.0), 32>>>(d_A, d_B, d_C, hA, wA, wB);
    else if (opt == 2)
        kernel_b<<<ceil(wB / 32.0), 32>>>(d_A, d_B, d_C, hA, wA, wB);
    else
        kernel_c<<<dim3(ceil(hA / 32.0), ceil(wB / 32.0)), dim3(32, 32)>>>(d_A, d_B, d_C, hA, wA, wB);

    hipMemcpy(C, d_C, hA * wB * size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Resultant array:\n");
    for (int i = 0; i < hA; i++)
    {
        for (int j = 0; j < wB; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }
}
int main()
{
    int hA = 2, wA = 3, wB = 3, option = 3;
    int A[hA][COL] = {{1, 2, 3}, {4, 5, 6}};
    int B[wA][COL] = {{5, 6, 9}, {15, 7, 15}, {2, 6, 4}};
    mul_matrix(A, B, hA, wA, wB, option);
    return 0;
}
// 1 2 3   5,6,9
// 4 5 6   15,7,15
//         2,6,4
