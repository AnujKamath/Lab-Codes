// Implement a CUDA program to add two vectors of length N by keeping the number of
// threads per block as 256 (constant) and vary the number of blocks to handle N elements.
#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void vecAddKernel(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
        c[i] = a[i] + b[i];
}
void vecAdd(int *a, int *b, int *c, int n)
{
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);
    hipMalloc((void **)&d_a, n * size);
    hipMalloc((void **)&d_b, n * size);
    hipMalloc((void **)&d_c, n * size);

    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(n / 256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);
    vecAddKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
int main()
{
    int n = 5;
    int a[n], b[n], c[n];
    for (int i = 0; i < n; i++)
    {
        a[i] = i * 2;
        b[i] = i * i;
    }

    for (int i = 0; i < n; i++)
        printf("%d ", a[i]);
    printf("\n");
    for (int i = 0; i < n; i++)
        printf("%d ", b[i]);
    printf("\n");

    vecAdd(a, b, c, n);
    printf("Result: ");

    for (int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");

    return 0;
}