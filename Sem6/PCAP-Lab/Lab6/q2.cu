#include "hip/hip_runtime.h"
// Write a program in CUDA to perform selection sort in parallel.
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void selection_sort_kernel(int *num, int *pos, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
    {
        for (int j = 0; j < n; j++)
            if (num[j] < num[i])
                pos[i]++;
    }
}
__global__ void place_kernel(int *num, int *ans, int *pos, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n)
    {
        ans[pos[i]] = num[i];
    }
}
void selection_sort(int *num, int *ans, int n)
{
    int pos[n] = {0};
    int *d_num, *d_pos, *d_ans;
    int size = sizeof(int);
    hipMalloc((void **)&d_num, n * size);
    hipMalloc((void **)&d_pos, n * size);
    hipMalloc((void **)&d_ans, n * size);

    hipMemcpy(d_num, num, n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, pos, n * size, hipMemcpyHostToDevice);

    dim3 dimGrid(n, 1, 1);
    dim3 dimBlock(1, 1, 1);
    selection_sort_kernel<<<dimGrid, dimBlock>>>(d_num, d_pos, n);

    place_kernel<<<dimGrid, dimBlock>>>(d_num, d_ans, d_pos, n);
    hipMemcpy(ans, d_ans, n * size, hipMemcpyDeviceToHost);

    hipFree(d_num);
    hipFree(d_pos);
    hipFree(d_ans);
}
int main()
{
    int n;
    printf("Enter size of array: ");
    scanf("%d", &n);
    int num[n], ans[n];
    printf("\nEnter values of array N: ");
    for (int i = 0; i < n; i++)
    {
        scanf("%d", &num[i]);
    }
    selection_sort(num, ans, n);
    printf("Sorted Array: ");
    for (int i = 0; i < n; i++)
        printf("%d ", ans[i]);
    printf("\n");

    return 0;
}
// Enter size of array: 5

// Enter values of array N: 3 2 1 5 4
// Sorted Array: 1 2 3 4 5