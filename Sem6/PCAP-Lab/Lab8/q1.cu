#include "hip/hip_runtime.h"
// Question 1. Write a progrm in CUDA to add two Matrices for the following specifications:
// a.Each row of resultant matrix to be computed by one thread.
// b.Each column of resultant matrix to be computed by one thread.
// c.Each element of resultant matrix to be computed by one thread.

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#define COL 3
__global__ void kernel_a(int *A, int *B, int *C, int r, int c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < r)
    {
        for (int k = 0; k < c; k++)
        {
            int x = A[i * c + k] + B[i * c + k];

            C[i * c + k] = x;
        }
    }
}
__global__ void kernel_b(int *A, int *B, int *C, int r, int c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < c)
    {
        for (int k = 0; k < r; k++)
        {
            C[c * k + i] = A[c * k + i] + B[c * k + i];
        }
    }
}
__global__ void kernel_c(int *A, int *B, int *C, int r, int c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < r && j < c)
    {
        C[i * c + j] = A[i * c + j] + B[i * c + j];
    }
}
void add_matrix(int A[][COL], int B[][COL], int r, int c, int opt)
{
    int C[r][c], size = sizeof(int);
    int *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, r * c * size);
    hipMalloc((void **)&d_B, r * c * size);
    hipMalloc((void **)&d_C, r * c * size);

    hipMemcpy(d_A, A, r * c * size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, r * c * size, hipMemcpyHostToDevice);

    if (opt == 1)
        kernel_a<<<ceil(r / 32.0), 32>>>(d_A, d_B, d_C, r, c);
    else if (opt == 2)
        kernel_b<<<ceil(c / 32.0), 32>>>(d_A, d_B, d_C, r, c);
    else
        kernel_c<<<dim3(ceil(r / 32.0), ceil(c / 32.0)), dim3(32, 32)>>>(d_A, d_B, d_C, r, c);

    hipMemcpy(C, d_C, r * c * size, hipMemcpyDeviceToHost);

    hipFree(d_A),
        hipFree(d_B);
    hipFree(d_C);
    printf("Resultant array:\n");
    for (int i = 0; i < r; i++)
    {
        for (int j = 0; j < c; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }
}
int main()
{
    int r = 2, c = 3, option = 3;
    int A[r][COL] = {{1, 2, 3}, {4, 5, 6}};
    int B[r][COL] = {{5, 6, 9}, {15, 7, 15}};
    add_matrix(A, B, r, c, option);
    return 0;
}
