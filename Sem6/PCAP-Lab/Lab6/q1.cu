// Question 1)Write a program in CUDA which performs convolution operation on one-dimensional input
// array N of size width using a mask array M of size mask_width to produce the resultant one-
// dimensional array P of size width.
#include "hip/hip_runtime.h"

#include <stdio.h>


__global__ void convolution_1D_basic_kernel(float *N, float *M, float *P, int Mask_Width, int width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width)
    {
        float Pvalue = 0;
        int N_start_point = i - (Mask_Width / 2);
        for (int j = 0; j < Mask_Width; j++)
            if (N_start_point + j >= 0 && N_start_point + j < width)
                Pvalue += N[N_start_point + j] * M[j];
        P[i] = Pvalue;
    }
}
void convolution_1D(float *N, float *M, float *P, int Mask_Width, int width)
{
    float *d_n, *d_m, *d_p;
    int size = sizeof(float);
    hipMalloc((void **)&d_n, width * size);
    hipMalloc((void **)&d_m, Mask_Width * size);
    hipMalloc((void **)&d_p, width * size);

    hipMemcpy(d_n, N, width * size, hipMemcpyHostToDevice);
    hipMemcpy(d_m, M, Mask_Width * size, hipMemcpyHostToDevice);
    hipMemcpy(d_p, P, width * size, hipMemcpyHostToDevice);

    float div = 1;
    dim3 dimGrid(ceil(width / div), 1, 1);
    dim3 dimBlock(div, 1, 1);
    convolution_1D_basic_kernel<<<dimGrid, dimBlock>>>(d_n, d_m, d_p, Mask_Width, width);

    hipMemcpy(P, d_p, width * size, hipMemcpyDeviceToHost);

    hipFree(d_n);
    hipFree(d_m);
    hipFree(d_p);
}

int main()
{
    int m_width, width, i;
    printf("Enter size of array and Mask: ");
    scanf("%d", &width);
    scanf("%d", &m_width);
    float N[width], M[m_width], P[width];

    fflush(stdin);
    printf("\nEnter values of array N: ");
    fflush(stdin);

    for (int k = 0; k < width; k++)
    {
        scanf("%f", &N[k]);
    }

    printf("\nEnter values of mask array:");
    for (i = 0; i < m_width; i++)
        scanf("%f", &M[i]);

    convolution_1D(N, M, P, m_width, width);
    printf("Result: ");

    for (i = 0; i < width; i++)
        printf("%.2f ", P[i]);
    printf("\n");

    return 0;
}
// Enter size of array and Mask: 7 5

// Enter values of array N: 1 2 3 4 5 6 7

// Enter values of mask array:3 4 5 4 3
// Result: 22.00 38.00 57.00 76.00 95.00 90.00 74.00