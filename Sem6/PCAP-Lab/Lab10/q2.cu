// Write a program in CUDA to improve the performance of 1D parallel convolution using constant
// Memory.
#include "hip/hip_runtime.h"

#include <stdio.h>


__constant__ float d_M_constant[256];

__global__ void convolution_1D_constant_kernel(float *N, float *P, int Mask_Width, int width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width)
    {
        float Pvalue = 0;
        int N_start_point = i - (Mask_Width / 2);
        for (int j = 0; j < Mask_Width; j++)
        {
            if (N_start_point + j >= 0 && N_start_point + j < width)
            {
                Pvalue += N[N_start_point + j] * d_M_constant[j];
            }
        }
        P[i] = Pvalue;
    }
}

void convolution_1D(float *N, float *M, float *P, int Mask_Width, int width)
{
    float *d_n, *d_p;
    int size = sizeof(float);
    hipMalloc((void **)&d_n, width * size);
    hipMalloc((void **)&d_p, width * size);

    hipMemcpy(d_n, N, width * size, hipMemcpyHostToDevice);
    hipMemcpy(d_p, P, width * size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(d_M_constant), M, Mask_Width * size);

    dim3 dimGrid(ceil(width / 32.0), 1);
    dim3 dimBlock(32, 1);

    convolution_1D_constant_kernel<<<dimGrid, dimBlock>>>(d_n, d_p, Mask_Width, width);

    hipMemcpy(P, d_p, width * size, hipMemcpyDeviceToHost);

    hipFree(d_n);
    hipFree(d_p);
}

int main()
{
    int m_width, width, i;
    printf("Enter size of array and Mask: ");
    scanf("%d", &width);
    scanf("%d", &m_width);
    float N[width], M[m_width], P[width];

    fflush(stdin);
    printf("\nEnter values of array N: ");
    fflush(stdin);

    for (int k = 0; k < width; k++)
    {
        scanf("%f", &N[k]);
    }

    printf("\nEnter values of mask array:");
    for (i = 0; i < m_width; i++)
        scanf("%f", &M[i]);

    convolution_1D(N, M, P, m_width, width);
    printf("Result: ");

    for (i = 0; i < width; i++)
        printf("%.2f ", P[i]);
    printf("\n");

    return 0;
}