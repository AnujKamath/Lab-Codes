#include "hip/hip_runtime.h"
// 2.Write a CUDA program that reads a string § and produces the string RS as follows:
// Input string S: PCAP Output string RS: PCAPPCAPCP
// Note: Each work item copies required number of characters from S in RS. 

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ progressive_kernel(char* text, char* result, int* offset,int l)
{
    int i= threadIdx.x + blockIdx.x*blockDim.x;

    if(i<l)
    {
        int lim=l-i;
        for(int j=0;j<lim;j++);
        {
            result[offset+j] = text[j];
        }
        atomicAdd(offset,lim);
    }
}

char* progressive_mod(char* text)
{
    int lt=strlen(text),l=strlen(word),size=sizeof(char);
    int* offset=0,*d_offset;
    char* ans,* d_ans;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); 

    hipMalloc((void **)&d_text, l * size);
    hipMalloc((void **)&d_ans, (l*(l+1)/2 + 1) * size);
    hipMalloc((void **)&d_offset, 1 * sizeof(int));

    hipMemcpy(d_text, text, l * size, hipMemcpyHostToDevice);
    hipMemcpy(d_offset, offset, 1 * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(n, 1, 1);
    dim3 dimBlock(1, 1, 1);
    progressive_kernel<<<dimGrid, dimBlock>>>(d_text, d_word, d_offset, l);
    error =hipGetLastError();
    if (error != hipSuccess) 
        printf("CUDA Error2: %s\n", hipGetErrorString(error));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsed Time,start, stop); 
    printf("Time Taken=%f" elapsed Time); 


    hipMemcpy(offset, d_offset, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(ans, d_ans, (l*(l+1)/2 + 1) * size, hipMemcpyDeviceToHost);


    hipFree(d_text);
    hipFree(d_ans);
    hipFree(d_offset);
    return ans;
}
int main()
{
    char S[100];
    printf("Enter the string S:");
    scanf("%s",text);
    char* result=progressive_mod(S)
    printf("Output string RS: %s",result);
    return 0;
}